#include "gpuOperationInterface.cuh"

//For error checking
void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS API failed!" << std::endl;
        exit(EXIT_FAILURE);
    }
}

//For error checking
void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        std::cerr << "CUDA API failed! " << hipGetErrorString(status) << std::endl;
        exit(EXIT_FAILURE);
    }
}



/*
Uses cublas to calculate the dotproduct between two matricies
*/
void cublasGpuDotProduct(CudaMemoryPool* memPool, double* matrixA, int matrixAHeight, int matrixAWidth, double* matrixB, int matrixBHeight, int matrixBWidth, double* matrixC, bool transposeA, bool transposeB){
    hipblasHandle_t handle;

    hipblasCreate(&handle);

    int matrixCHeight = matrixAHeight;
    if(transposeA){
        matrixCHeight = matrixAWidth;
    }

    int matrixCWidth = matrixBWidth;
    if(transposeB){
        matrixCWidth = matrixBHeight;
    }


    size_t matrixABytes = matrixAWidth * matrixAHeight * sizeof(double);
    size_t matrixBBytes = matrixBWidth * matrixBHeight * sizeof(double);
    size_t matrixCBytes = matrixCWidth * matrixCHeight * sizeof(double);

    double *cudaMatrixA, *cudaMatrixB, *cudaMatrixC;
    /*
    checkCudaStatus(hipMallocManaged(&cudaMatrixA, matrixABytes));
    checkCudaStatus(hipMallocManaged(&cudaMatrixB, matrixBBytes));
    checkCudaStatus(hipMallocManaged(&cudaMatrixC, matrixCBytes));
    */

    cudaMatrixA = memPool->cudaRequestMemory(matrixABytes);
    cudaMatrixB = memPool->cudaRequestMemory(matrixBBytes);
    cudaMatrixC = memPool->cudaRequestMemory(matrixCBytes);

    //Transfering the matricies to the device (gpu), this also changes the format to column leading
    for(int i = 0; i<matrixAHeight; i++){
        for(int j = 0; j<matrixAWidth; j++){
            cudaMatrixA[ACCESSCOLLEADING2D(i, j, matrixAHeight)] = matrixA[ACCESSROWLEADING2D(i, j, matrixAWidth)];
            //cout<<matrixA[ACCESSROWLEADING2D(i, j, matrixAWidth)]<<", ";
        }
        //cout<<endl;
    }

    for(int i = 0; i<matrixBHeight; i++){
        for(int j = 0; j<matrixBWidth; j++){
            cudaMatrixB[ACCESSCOLLEADING2D(i, j, matrixBHeight)] = matrixB[ACCESSROWLEADING2D(i, j, matrixBWidth)];
            //cout<<matrixB[ACCESSROWLEADING2D(i, j, matrixBWidth)]<<", ";
        }
        //cout<<endl;
    }


    // Storing values needed to perform dot product
    double alpha = 1.0;
    double beta = 0.0;

    int lda = matrixAHeight;
    int ldb = matrixBHeight;
    int ldc = matrixCHeight;

    if(transposeA){
        int tempHeight = matrixAHeight;
        matrixAHeight = matrixAWidth;
        matrixAWidth = tempHeight;
    }

    if(transposeB){
        int tempHeight = matrixBHeight;
        matrixBHeight = matrixBWidth;
        matrixBWidth = tempHeight;
    }

    //This is the cublas matrix multiplication algorithm
    checkCublasStatus(hipblasDgemm(handle, (hipblasOperation_t)transposeA, (hipblasOperation_t)transposeB, matrixAHeight, matrixBWidth, matrixAWidth, &alpha, cudaMatrixA, lda, cudaMatrixB, ldb, &beta, cudaMatrixC, ldc));

    //Wait for the threads to complete
    checkCudaStatus(hipDeviceSynchronize());

    //Copy the data back into the matrix C array
    for(int i = 0; i<matrixCHeight; i++){
        for(int j = 0; j<matrixCWidth; j++){
            matrixC[ACCESSROWLEADING2D(i, j, matrixCWidth)] = cudaMatrixC[ACCESSCOLLEADING2D(i, j, matrixCHeight)];
        }
    }

    //Free memory
    /*
    hipFree(cudaMatrixA);
    hipFree(cudaMatrixB);
    hipFree(cudaMatrixC);
    */
    memPool->unreserveMemory(cudaMatrixA);
    memPool->unreserveMemory(cudaMatrixB);
    memPool->unreserveMemory(cudaMatrixC);
    hipblasDestroy(handle);
}


CudaMemoryPool::CudaMemoryPool(){
    this->memoryPoolSize = 0;
}

CudaMemoryPool::~CudaMemoryPool(){
    for(int i = 0; i<memoryPoolSize; i++){
        //Check that the memory is not currently being used
        assert(get<2>(this->memoryPoolArray.at(i)) == false);
        //Unallocate all memory
        checkCudaStatus(hipFree(get<0>(this->memoryPoolArray.at(i))));
    }
}

double* CudaMemoryPool::cudaRequestMemory(size_t requestedSize){
    //Loop through all potential memory blocks
    for(int i = 0; i<memoryPoolSize; i++){
        //Check if its being used
        if(get<2>(this->memoryPoolArray.at(i)) == false){
            get<2>(this->memoryPoolArray.at(i)) = true;
            //Resize if its too small
            if(get<1>(this->memoryPoolArray.at(i)) < requestedSize){
                checkCudaStatus(hipFree(get<0>(this->memoryPoolArray.at(i))));
                checkCudaStatus(hipMallocManaged(&get<0>(this->memoryPoolArray.at(i)), requestedSize));
            }
            return get<0>(this->memoryPoolArray.at(i));
        }
    }

    //Create a new memory block
    memoryPoolSize++;
    assert(memoryPoolSize<=100);

    checkCudaStatus(hipMallocManaged(&get<0>(this->memoryPoolArray.at(memoryPoolSize-1)), requestedSize));
    get<1>(memoryPoolArray.at(memoryPoolSize-1)) = requestedSize;
    get<2>(memoryPoolArray.at(memoryPoolSize-1)) = true;
    return get<0>(this->memoryPoolArray.at(memoryPoolSize-1));
}

void CudaMemoryPool::unreserveMemory(double* memoryAdr){
    //Loop through all potential memory blocks
    for(int i = 0; i<memoryPoolSize; i++){
        if(get<0>(memoryPoolArray.at(i)) == memoryAdr){
            get<2>(memoryPoolArray.at(i)) = false;
            return;
        }
    }
    cout<<"Memory does not match any in pool!"<<endl;
    assert(false);
}

//nvcc -o gpuOperationInterface gpuOperationInterface.cu -lcublas -lcuda


/*
//Main for testing
int main (void){

    int matrixAHeight = 3;
    int matrixAWidth = 2;
    int matrixBHeight = 3; 
    int matrixBWidth = 3;
    int matrixCHeight = 2;
    int matrixCWidth = 3;

    size_t matrixABytes = matrixAHeight * matrixAWidth * sizeof(double);
    size_t matrixBBytes = matrixBHeight * matrixBWidth * sizeof(double);
    size_t matrixCBytes = matrixCHeight * matrixCWidth * sizeof(double);

    double* matrixA = (double*)malloc(matrixABytes);
    double* matrixB = (double*)malloc(matrixBBytes);
    double* matrixC = (double*)malloc(matrixCBytes);


    
    for(int i = 0; i<matrixAHeight*matrixAWidth; i++){
        matrixA[i] = i;
    }
    for(int i = 0; i<matrixBHeight*matrixBWidth; i++){
        matrixB[i] = i;
    }


    cublasGpuDotProduct(matrixA, matrixAHeight, matrixAWidth, matrixB, matrixBHeight, matrixBWidth, matrixC, true, true);

    for(int i = 0; i<matrixCHeight; i++){
        for(int j = 0; j<matrixCWidth; j++){
            cout<<matrixC[ACCESSROWLEADING2D(i, j, matrixCWidth)]<<", ";
        }
        cout<<endl;
    }

    free(matrixA);
    free(matrixB);
    free(matrixC);   
}
*/