#include "hip/hip_runtime.h"
#include "gpuOperationInterface.cuh"

//For error checking
void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS API failed!" << std::endl;
        exit(EXIT_FAILURE);
    }
}

//For error checking
void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        std::cerr << "CUDA API failed! " << hipGetErrorString(status) << std::endl;
        exit(EXIT_FAILURE);
    }
}



/*
Uses cublas to calculate the dotproduct between two matricies
*/

void cublasGpuDotProduct(double* cudaMatrixA, int matrixAHeight, int matrixAWidth, double* cudaMatrixB, int matrixBHeight, int matrixBWidth, double* cudaMatrixC, bool transposeA, bool transposeB){
    //TODO: Store this handle somewhere so we dont have to keep deleting it (probs in comp graph)
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int matrixCHeight = matrixAHeight;
    if(transposeA){
        matrixCHeight = matrixAWidth;
    }

    int matrixCWidth = matrixBWidth;
    if(transposeB){
        matrixCWidth = matrixBHeight;
    }



    // Storing values needed to perform dot product
    double alpha = 1.0;
    double beta = 0.0;

    int lda = matrixAHeight;
    int ldb = matrixBHeight;
    int ldc = matrixCHeight;

    if(transposeA){
        int tempHeight = matrixAHeight;
        matrixAHeight = matrixAWidth;
        matrixAWidth = tempHeight;
    }

    if(transposeB){
        int tempHeight = matrixBHeight;
        matrixBHeight = matrixBWidth;
        matrixBWidth = tempHeight;
    }

    //This is the cublas matrix multiplication algorithm
    checkCublasStatus(hipblasDgemm(handle, (hipblasOperation_t)transposeA, (hipblasOperation_t)transposeB, matrixAHeight, matrixBWidth, matrixAWidth, &alpha, cudaMatrixA, lda, cudaMatrixB, ldb, &beta, cudaMatrixC, ldc));

    //Wait for the threads to complete
    checkCudaStatus(hipDeviceSynchronize());

    hipblasDestroy(handle);
}




void cublasGpuDotProductOld(CudaMemoryPool* memPool, double* matrixA, int matrixAHeight, int matrixAWidth, double* matrixB, int matrixBHeight, int matrixBWidth, double* matrixC, bool transposeA, bool transposeB){
    hipblasHandle_t handle;

    hipblasCreate(&handle);

    int matrixCHeight = matrixAHeight;
    if(transposeA){
        matrixCHeight = matrixAWidth;
    }

    int matrixCWidth = matrixBWidth;
    if(transposeB){
        matrixCWidth = matrixBHeight;
    }


    size_t matrixABytes = matrixAWidth * matrixAHeight * sizeof(double);
    size_t matrixBBytes = matrixBWidth * matrixBHeight * sizeof(double);
    size_t matrixCBytes = matrixCWidth * matrixCHeight * sizeof(double);

    double *cudaMatrixA, *cudaMatrixB, *cudaMatrixC;
    /*
    checkCudaStatus(hipMallocManaged(&cudaMatrixA, matrixABytes));
    checkCudaStatus(hipMallocManaged(&cudaMatrixB, matrixBBytes));
    checkCudaStatus(hipMallocManaged(&cudaMatrixC, matrixCBytes));
    */

    cudaMatrixA = memPool->cudaRequestMemory(matrixABytes);
    cudaMatrixB = memPool->cudaRequestMemory(matrixBBytes);
    cudaMatrixC = memPool->cudaRequestMemory(matrixCBytes);

    //Transfering the matricies to the device (gpu), this also changes the format to column leading
    for(int i = 0; i<matrixAHeight; i++){
        for(int j = 0; j<matrixAWidth; j++){
            cudaMatrixA[ACCESSCOLLEADING2D(i, j, matrixAHeight)] = matrixA[ACCESSCOLLEADING2D(i, j, matrixAHeight)];
            //cout<<matrixA[ACCESSROWLEADING2D(i, j, matrixAWidth)]<<", ";
        }
        //cout<<endl;
    }

    for(int i = 0; i<matrixBHeight; i++){
        for(int j = 0; j<matrixBWidth; j++){
            cudaMatrixB[ACCESSCOLLEADING2D(i, j, matrixBHeight)] = matrixB[ACCESSCOLLEADING2D(i, j, matrixBHeight)];
            //cout<<matrixB[ACCESSROWLEADING2D(i, j, matrixBWidth)]<<", ";
        }
        //cout<<endl;
    }


    // Storing values needed to perform dot product
    double alpha = 1.0;
    double beta = 0.0;

    int lda = matrixAHeight;
    int ldb = matrixBHeight;
    int ldc = matrixCHeight;

    if(transposeA){
        int tempHeight = matrixAHeight;
        matrixAHeight = matrixAWidth;
        matrixAWidth = tempHeight;
    }

    if(transposeB){
        int tempHeight = matrixBHeight;
        matrixBHeight = matrixBWidth;
        matrixBWidth = tempHeight;
    }

    //This is the cublas matrix multiplication algorithm
    checkCublasStatus(hipblasDgemm(handle, (hipblasOperation_t)transposeA, (hipblasOperation_t)transposeB, matrixAHeight, matrixBWidth, matrixAWidth, &alpha, cudaMatrixA, lda, cudaMatrixB, ldb, &beta, cudaMatrixC, ldc));

    //Wait for the threads to complete
    checkCudaStatus(hipDeviceSynchronize());

    //Copy the data back into the matrix C array
    for(int i = 0; i<matrixCHeight; i++){
        for(int j = 0; j<matrixCWidth; j++){
            matrixC[ACCESSCOLLEADING2D(i, j, matrixCHeight)] = cudaMatrixC[ACCESSCOLLEADING2D(i, j, matrixCHeight)];
        }
    }

    //Free memory
    /*
    hipFree(cudaMatrixA);
    hipFree(cudaMatrixB);
    hipFree(cudaMatrixC);
    */
    memPool->unreserveMemory(cudaMatrixA);
    memPool->unreserveMemory(cudaMatrixB);
    memPool->unreserveMemory(cudaMatrixC);
    hipblasDestroy(handle);
}

__global__
void deviceGpuEncode(double* cudaIndexes, int indexesHeight, int indexesWidth, double* cudaEmbeddingTable, int embeddingHeight, int embeddingWidth, double* cudaOutMatrix, int outHeight, int outWidth){
    //Because we set up the blocks and threads to match with input matrix we can access the current row by looking at the y dimention
    int currentRow = (blockIdx.y * blockDim.y) + threadIdx.y;
    //The same can be done to get the current collumn by accessing the x dimention
    int currentCol = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(currentRow < indexesHeight && currentCol < indexesWidth){
        //Define the current index we are accessing
        int currentIndex = cudaIndexes[ACCESSCOLLEADING2D(currentRow, currentCol, indexesHeight)];
        for(int i = 0; i<embeddingWidth; i++){
            cudaOutMatrix[ACCESSCOLLEADING2D(currentRow, (currentCol * embeddingWidth) + i, outHeight)] =
                cudaEmbeddingTable[ACCESSCOLLEADING2D(currentIndex, i, embeddingHeight)];
        }
    }
}




CudaMemoryPool::CudaMemoryPool(){
    this->memoryPoolSize = 0;
}

CudaMemoryPool::~CudaMemoryPool(){
    for(int i = 0; i<memoryPoolSize; i++){
        //Check that the memory is not currently being used
        assert(get<2>(this->memoryPoolArray.at(i)) == false);
        //Unallocate all memory
        checkCudaStatus(hipFree(get<0>(this->memoryPoolArray.at(i))));
    }
}

double* CudaMemoryPool::cudaRequestMemory(size_t requestedSize){
    //Loop through all potential memory blocks
    for(int i = 0; i<memoryPoolSize; i++){
        //Check if its being used
        if(get<2>(this->memoryPoolArray.at(i)) == false){
            get<2>(this->memoryPoolArray.at(i)) = true;
            //Resize if its too small
            if(get<1>(this->memoryPoolArray.at(i)) < requestedSize){
                checkCudaStatus(hipFree(get<0>(this->memoryPoolArray.at(i))));
                checkCudaStatus(hipMallocManaged(&get<0>(this->memoryPoolArray.at(i)), requestedSize));
            }
            return get<0>(this->memoryPoolArray.at(i));
        }
    }

    //Create a new memory block
    memoryPoolSize++;
    assert(memoryPoolSize<=100);

    checkCudaStatus(hipMallocManaged(&get<0>(this->memoryPoolArray.at(memoryPoolSize-1)), requestedSize));
    get<1>(memoryPoolArray.at(memoryPoolSize-1)) = requestedSize;
    get<2>(memoryPoolArray.at(memoryPoolSize-1)) = true;
    return get<0>(this->memoryPoolArray.at(memoryPoolSize-1));
}

void CudaMemoryPool::unreserveMemory(double* memoryAdr){
    //Loop through all potential memory blocks
    for(int i = 0; i<memoryPoolSize; i++){
        if(get<0>(memoryPoolArray.at(i)) == memoryAdr){
            get<2>(memoryPoolArray.at(i)) = false;
            return;
        }
    }
    cout<<"Memory does not match any in pool!"<<endl;
    assert(false);
}
