#include "CudaMemoryClass.cuh"


CudaMemoryClass::CudaMemoryClass(){
    this->currentAllocatedMemory = 0;
    this->currentUsedMemory = 0;
    this->cudaMemPtr = nullptr;
}

CudaMemoryClass::~CudaMemoryClass(){
    checkCudaStatus(hipFree(this->cudaMemPtr));
}

double* CudaMemoryClass::setUsedMemory(size_t memSize){
    if(memSize>currentAllocatedMemory){
        
        double* newMemory;
        checkCudaStatus(hipMallocManaged(&newMemory, memSize));
        if(this->cudaMemPtr != nullptr){
            checkCudaStatus(hipMemcpy(newMemory ,this->cudaMemPtr, this->currentUsedMemory, hipMemcpyDeviceToDevice));
            checkCudaStatus(hipFree(this->cudaMemPtr)); 
        }
        this->cudaMemPtr = newMemory;
        this->currentAllocatedMemory = memSize;
        
    }
    this->currentUsedMemory = memSize;

    return this->cudaMemPtr;
    
}

void CudaMemoryClass::transferToHost(double* dest){
    checkCudaStatus(hipMemcpy(dest, this->cudaMemPtr, this->currentUsedMemory, hipMemcpyDeviceToHost));
}

void CudaMemoryClass::transferToDevice(double* src){
    checkCudaStatus(hipMemcpy(this->cudaMemPtr, src, this->currentUsedMemory, hipMemcpyHostToDevice));
}